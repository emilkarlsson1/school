/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>


static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {

    return (a + b - 1)/b;
}

__global__ void correlate_kernel(int num_rows, int num_cols, const float* data, float* result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= num_rows || j >= num_rows)
        return;

    float sum = 0;
    for (int k = 0; k < num_cols; k++){
        sum += data[k + i * num_cols] * data[k + j * num_cols];
    }
    result[j + i * num_rows] = sum;
}

void correlate(int ny, int nx, const float *data, float *result) {

    float *normalized_data = (float *)malloc(sizeof(float) * ny * nx);
    for (int i = 0; i < ny; i ++) {
        float row_sum = 0;
        
        for (int j = 0; j < nx; j ++) {
            row_sum += data[j + i * nx];
        }
        float row_mean = row_sum / nx;

        float row_square_sum = 0;
        
        for (int j = 0; j < nx; j ++) {
            float x = data[j + i * nx] - row_mean;
            normalized_data[j + i * nx] = x;
            row_square_sum += x * x;
        }

        row_square_sum = sqrt(row_square_sum);
        for(int j = 0; j < nx; j++) {
            normalized_data[j + i * nx] /= row_square_sum;
        }
    }

 
    float* d_norm_data = NULL;
    float* d_result = NULL;
    CHECK(hipMalloc((void**)&d_norm_data, ny * nx * sizeof(float)));
    CHECK(hipMalloc((void**)&d_result, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(d_norm_data, normalized_data, ny * nx * sizeof(float), hipMemcpyHostToDevice));


    dim3 block_size(32, 32);
    dim3 grid_size(divup(ny, block_size.x), divup(ny, block_size.y));
    correlate_kernel<<<grid_size, block_size>>>(ny, nx, d_norm_data, d_result);
    CHECK(hipGetLastError());


    CHECK(hipMemcpy(result, d_result, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_norm_data));
    CHECK(hipFree(d_result));

    free(normalized_data);
}
