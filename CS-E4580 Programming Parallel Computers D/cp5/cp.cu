/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

//from slides
__global__ void correlate_kernel(int nx, int ny, int nx_p, int ny_p, float* transposed, float* result) {
   
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;

    if (ic > jc) {
        for (int ib = 0; ib < 8; ib++) {
            for (int jb = 0; jb < 8; jb++) {
                int i = ic * 64 + ib * 8 + ia;
                int j = jc * 64 + jb * 8 + ja;
                if (i < ny && j < ny) {
                	result[j + i*ny] = 0.0;
                }
            }
        }
    } else {
	    float v[8][8];
	    for (int ib = 0; ib < 8; ib++) {
		for (int jb = 0; jb < 8; jb++) {
		    v[ib][jb] = 0;
		}
	    }
	    
	    for (int k = 0; k < nx; k++) {
		float x[8];
		float y[8];
		for (int ib = 0; ib < 8; ib++) {
		    int i = ic * 64 + ib * 8 + ia;
		    x[ib] = transposed[ny_p * k + i];
		}
		
		for (int jb = 0; jb < 8; jb++) {
		    int j = jc * 64 + jb * 8 + ja;
		    y[jb] = transposed[ny_p * k + j];
		}
		
		for (int ib = 0; ib < 8; ib++) {
		    for (int jb = 0; jb < 8; jb++) {
		        v[ib][jb] += x[ib] * y[jb];
		    }
		}
	    }
	    
	    for (int ib = 0; ib < 8; ib++) {
		for (int jb = 0; jb < 8; jb++) {
		    int i = ic * 64 + ib * 8 + ia;
		    int j = jc * 64 + jb * 8 + ja;
		    if (i < ny && j < ny) {
		    	result[j + i*ny] = v[ib][jb];
		    }
		}
	    }
    }
}

__global__ void correlate_kernel_transpose(int nx, int ny, int nx_p, int ny_p, float* transposed, float* normalized) {
    int ja = threadIdx.x; 
    int i = blockIdx.y; 

    for (int jb = 0; jb < nx_p; jb += 64) 
    {
        int j = jb + ja;
        transposed[ny_p * j + i] = (i < ny && j < nx) ? normalized[nx * i + j] : 0;
    }
}

__global__ void gpu_normalize(int nx, int ny, int nx_p, int ny_p, float* normalized, float* data) {

    int ja = threadIdx.y;
    int i = blockIdx.y;

    int y = i * 64 + ja;
    if (y < ny) {
        float row_sum = 0.0;
        float row_square_sum = 0.0;
        
        for (int x = 0; x < nx; x++) {   
            row_sum += data[x + y*nx];
        }
        
        float rwo_avg = row_sum/nx;
        for (int x = 0; x < nx; x++) {   
            float item = data[x + y*nx]-rwo_avg;
            normalized[x + y*nx] = item;
            row_square_sum += pow(item, 2);
        }
        
        float root_square_sum = sqrt(row_square_sum);
        for (int x = 0; x < nx; x++) {
            normalized[x + y*nx] /= root_square_sum;         
        }
    }
}


void correlate(int ny, int nx, const float* data, float* result) {
    
    int nx_p = roundup(nx, 64);
    int ny_p = roundup(ny, 64);

    //initialize pointers to null
    float* dGPU = NULL;
    float* dGPU_raw = NULL;
    float* dGPU_norm = NULL;
    float* rGPU = NULL;


    const int input_sz = nx_p * ny_p * sizeof(float);
    const int mtx_sz = nx * ny * sizeof(float);
    const int out_sz = ny * ny * sizeof(float);


    CHECK(hipMalloc((void**)&dGPU, input_sz));
    CHECK(hipMalloc((void**)&dGPU_raw, mtx_sz));
    CHECK(hipMalloc((void**)&dGPU_norm, mtx_sz));
    CHECK(hipMalloc((void**)&rGPU, out_sz));
    CHECK(hipMemcpy(dGPU_raw, data, mtx_sz, hipMemcpyHostToDevice));

  
    {
        dim3 dimBlock(1, 64);
        dim3 dimGrid(1, ny_p/64);
        gpu_normalize<<<dimGrid, dimBlock>>>(nx, ny, nx_p, ny_p, dGPU_norm, dGPU_raw);
        CHECK(hipGetLastError());
    }

    {
        dim3 dimBlock(64, 1);   
        dim3 dimGrid(1, ny_p);  
        correlate_kernel_transpose<<<dimGrid, dimBlock>>>(nx, ny, nx_p, ny_p, dGPU, dGPU_norm);
        CHECK(hipGetLastError());
    }

    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(ny_p / 64, ny_p / 64); 
        correlate_kernel<<<dimGrid, dimBlock>>>(nx, ny, nx_p, ny_p, dGPU, rGPU);
        CHECK(hipGetLastError());
    }


    CHECK(hipMemcpy(result, rGPU, out_sz, hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(dGPU_raw));
    CHECK(hipFree(dGPU_norm));
    CHECK(hipFree(rGPU));
}
